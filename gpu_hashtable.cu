#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>

#include "gpu_hashtable.hpp"

#define BLOCKSIZE 1024
#define BLOCKNUM 1
#define A 113

typedef struct {
	int state; // 0 -> unoccupied | 1 -> occupied | -1 -> was previously
	int key;
	int value;
} Pair;

typedef struct {
	int *size;
	int *numElem;
	Pair *pairs;
} HashTable;

HashTable hashTable;

/* INIT HASH
 */
GpuHashTable::GpuHashTable(int size) {
	hipMalloc(&hashTable.size, sizeof(int));
	hipMalloc(&hashTable.pairs, size * sizeof(Pair));
	hipMalloc(&hashTable.numElem, sizeof(int));

	hipMemset(hashTable.numElem, 0, sizeof(int));
	hipMemset(hashTable.pairs, 0, size * sizeof(Pair));
	hipMemcpy(hashTable.size, &size, sizeof(int), hipMemcpyHostToDevice);
	
	//printf("SIZE: %d\n", size);
}

void print_hash() {
	printf("SE AFISEAZA HASHTABLE\n");

	Pair *hostPairs;

	int *hostNumElem = (int *)malloc(sizeof(int));
	hipMemcpy(hostNumElem, hashTable.numElem, sizeof(int), hipMemcpyDeviceToHost);
	printf("NUMAR DE ELEMENTE: %d\n", *hostNumElem);

	int *hostSize = (int *)malloc(sizeof(int));
	hipMemcpy(hostSize, hashTable.size, sizeof(int), hipMemcpyDeviceToHost);
	printf("HASHTABLE SIZE: %d\n", *hostSize);


	hostPairs = (Pair *)malloc (sizeof(Pair) * (*hostSize));

	hipMemcpy(hostPairs, hashTable.pairs, sizeof(Pair) * (*hostSize), hipMemcpyDeviceToHost);
	for (int i = 0; i < (*hostSize); i++) {
	//	if (hostPairs[i].key != 0) {
			printf("%d. Key: %d Value: %d \n", i, hostPairs[i].key, hostPairs[i].value);
	//	}
	}

	printf("\n");
}

/* DESTROY HASH
 */
GpuHashTable::~GpuHashTable() {
	hipFree(hashTable.pairs);
	hipFree(hashTable.size);
	hipFree(hashTable.numElem);
}

__global__ void resize(Pair *newPairs, Pair *oldPairs, int *size, int *numElem, int *oldSize) {
	int keyToInsert = blockIdx.x * blockDim.x + threadIdx.x;
	if (keyToInsert >= *oldSize)
		return;

	int key = oldPairs[keyToInsert].key;
	if (key == 0)
		return;

	int position = hash1(key, *size);
	int index = position;
	int free = 0;
	/*
	while(1) {
		//if (pairs[index].key == key) 
		//	break;

		if (atomicCAS(&(pairs[index].key), free, key) == 0) {
			atomicAdd(result, 1);
			atomicAdd(numElem, 1);
			break;
		} */
	while (atomicCAS(&(newPairs[index].key), free, key) != 0) {
		index++;
		if (index == (*size))
			index = 0;
	}
	
	atomicAdd(numElem, 1);
	
	newPairs[index].value = oldPairs[keyToInsert].value;
}


/* RESHAPE HASH
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	//int hostNumElem;
	//int hostOldSize;
	//int *hostKeys, *hostValues;
	//Pair *hostPairs;

	//TODO: check the allocations
	printf("\n");
	printf("FAC RESHAPE CU %d\n", numBucketsReshape);

	/* Get old size of hashTable */
	//hipMemcpy(&hostOldSize, hashTable.size, sizeof(int), hipMemcpyDeviceToHost);

	/* Get current number of elements */
	//hipMemcpy(&hostNumElem, hashTable.numElem, sizeof(int), hipMemcpyDeviceToHost);

	/* Get all pairs in hashTable */
	//hostPairs = (Pair *) malloc(sizeof(Pair) * hostOldSize);
	//hipMemcpy(hostPairs, hashTable.pairs, sizeof(Pair) * hostOldSize, hipMemcpyDeviceToHost);

//	hostKeys = (int *) malloc(sizeof(int) * hostOldSize);
//	hostValues = (int *) malloc(sizeof(int) * hostOldSize);

	/* Filter pairs*/
/*	int index = 0;
	for (int i = 0; i < hostOldSize; i++) {
		if (hostPairs[i].key != 0) {
			hostKeys[index] = hostPairs[i].key;
			hostValues[index] = hostPairs[i].value;
			index++;
		}
	}
*/	
//	printf("test:\n");
//	for (int i = 0; i < index; i++) {
//		printf("key %d value %d\n", hostKeys[i], hostValues[i]);
//	}

	Pair *devicePairs;
	int hostOldSize;
	int *deviceOldSize;

	hipMalloc(&deviceOldSize, sizeof(int));
	hipMalloc(&devicePairs, numBucketsReshape * sizeof(Pair));

	hipMemcpy(deviceOldSize, hashTable.size, sizeof(int), hipMemcpyHostToHost);
	hipMemcpy(&hostOldSize, hashTable.size, sizeof(int), hipMemcpyHostToHost);

	hipMemset(hashTable.numElem, 0, sizeof(int));
	hipMemcpy(hashTable.size, &numBucketsReshape, sizeof(int), hipMemcpyHostToDevice);	

	int blockNum = hostOldSize / BLOCKSIZE;
	if (blockNum * BLOCKSIZE < hostOldSize)
		blockNum++;

	resize<<<blockNum, BLOCKSIZE>>>(devicePairs, hashTable.pairs,
			hashTable.size, hashTable.numElem, deviceOldSize);
	
	hipDeviceSynchronize();

	hipFree(hashTable.pairs);
	hashTable.pairs = devicePairs;

	//hashTable.pairs = devicePairs;

//	insertBatch(hostKeys, hostValues, hostNumElem);

//	print_hash();
//	free(hostPairs);
//	free(hostValues);
//	free(hostKeys);

	printf("\n");
}

__global__ void insert(int *keys, int *values, Pair *pairs, int *size,
		int *numElem, int *result, int numKeys) {
	/* Get position in HashTable for inertion */
	int keyToInsert = blockIdx.x * blockDim.x + threadIdx.x;
	if (keyToInsert >= numKeys)
		return;

	if (keys[keyToInsert] <= 0 || values[keyToInsert] <= 0)
		return;

//	int position = keys[keyToInsert] * A % (*size);
	int position = hash1(keys[keyToInsert], *size);
	int key = keys[keyToInsert];
	int index = position;
	int free = 0;

	/* Check for an empty space in the HashTable */
	/*while ((atomicCAS(&(pairs[index].key), free, key) != 0) &&
		(atomicCAS(&(pairs[index].key), key, key) != key)) {
		index++;
		if (index == (*size))
			index = 0;
	}  */
	while(1) {
	//	if (atomicCAS(&(pairs[index].key, key, key) == key))
	//		break;
		if (pairs[index].key == key) 
			break;

		if (atomicCAS(&(pairs[index].key), free, key) == 0) {
			atomicAdd(result, 1);
			atomicAdd(numElem, 1);
			break;
		}

		index++;
		if (index == (*size))
			index = 0;
	}
	
	//atomicAdd(result, 1);
	//atomicAdd(numElem, 1);
		
	//pairs[index].key = keys[keyToInsert];
	pairs[index].value = values[keyToInsert];
}

/* INSERT BATCH
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	int *deviceResult, *deviceKeys, *deviceValues;
	int *hostResult;
	bool returnValue = false;

	int *hostNumElem = (int *)malloc(sizeof(int));
	hipMemcpy(hostNumElem, hashTable.numElem, sizeof(int), hipMemcpyDeviceToHost);
//	printf("NUMAR DE ELEMENTE: %d\n", *hostNumElem);

	int *hostSize = (int *)malloc(sizeof(int));
	hipMemcpy(hostSize, hashTable.size, sizeof(int), hipMemcpyDeviceToHost);
//	printf("HASHTABLE SIZE: %d\n", *hostSize);

	int oldSize = (*hostSize);
	while ((numKeys + (*hostNumElem)) > (*hostSize)) {
		(*hostSize) *= 2;
	}

	if ((*hostNumElem) + numKeys > 0) { 
		if ((((float)((*hostNumElem) + numKeys) / (*hostSize)) > 0.8f) 
			&& ((*hostSize) * 2 < 0x01111111)) {
			(*hostSize) *= 2;
		}
	}

	if (oldSize != (*hostSize))
		reshape((*hostSize));

	//TODO: check the allocations
	hostResult = (int *) malloc(sizeof(int));
	*hostResult = 0;

	hipMalloc(&deviceKeys, numKeys * sizeof(int));
	hipMalloc(&deviceValues, numKeys * sizeof(int));
	hipMalloc(&deviceResult, sizeof(int));

	hipMemcpy(deviceKeys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceValues, values, numKeys * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceResult, hostResult, sizeof(int), hipMemcpyHostToDevice);

	int blockNum = numKeys / BLOCKSIZE;
	if (blockNum * BLOCKSIZE < numKeys)
		blockNum++;
	
	insert<<<blockNum, BLOCKSIZE>>>(deviceKeys, deviceValues, hashTable.pairs, 
					hashTable.size, hashTable.numElem, deviceResult, numKeys);
	
	hipDeviceSynchronize();

	hipFree(deviceResult);
	hipFree(deviceKeys);
	hipFree(deviceValues);
	free(hostResult);

//	printf("\n");
//	print_hash();
//	printf("\n");

	return returnValue;
}

__global__ void get(int *keys, int *values, Pair *pairs, int *size, int numKeys) {
	/* Get position in HashTable for inertion */
	int keyToGet = blockIdx.x * blockDim.x + threadIdx.x;
	if (keyToGet >= numKeys)
		return;

	int position = hash1(keys[keyToGet], *size);
	//int position = keys[keyToGet] * A % (*size);
	int key = keys[keyToGet];
	int index = position;
	int free = 0;
	int round = 0;

	/* Check for an empty space in the HashTable */
	while ((atomicCAS(&(pairs[index].key), key, key) != key) &&
		(atomicCAS(&(pairs[index].key), free, free) != free)) {
		index++;
		if (index == (*size))
			index = 0;

		if ((index == position) && (round == 1))
			return;

		round = 1;
	}

	values[keyToGet] = pairs[index].value;
}

/* GET BATCH
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
	int *deviceKeys, *deviceValues;
	int *hostValues;

	//TODO: check the allocations
	hostValues = (int *) calloc(numKeys, sizeof(int));

	hipMalloc(&deviceKeys, numKeys * sizeof(int));
	hipMalloc(&deviceValues, numKeys * sizeof(int));
	hipMemcpy(deviceKeys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
		
	int blockNum = numKeys / BLOCKSIZE;
	if (blockNum * BLOCKSIZE < numKeys)
		blockNum++;
	
	get<<<blockNum, BLOCKSIZE>>>(deviceKeys, deviceValues, hashTable.pairs, 
					hashTable.size, numKeys);
	
	hipDeviceSynchronize();
	hipMemcpy(hostValues, deviceValues, sizeof(int) * numKeys, hipMemcpyDeviceToHost);
	
	hipFree(deviceKeys);
	hipFree(deviceValues);
	
	return hostValues;
}

/* GET LOAD FACTOR
 * num elements / hash total slots elements
 */
float GpuHashTable::loadFactor() {
	int *numElem = (int *) malloc(sizeof(int));
	int *size = (int *) malloc(sizeof(int));

	hipMemcpy(numElem, hashTable.numElem, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(size, hashTable.size, sizeof(int), hipMemcpyDeviceToHost);
	
	float loadFactor = (float) *numElem / *size;
	
	return (float)loadFactor; // no larger than 1.0f = 100%
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#include "test_map.cpp"
